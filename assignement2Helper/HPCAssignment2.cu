#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <./helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#define MAX_EPSILON_ERROR 5e-3f
#define TILE_WIDTH 16
#define FILTERDIM 3                     //CHANGE THIS WHEN USING DIFFERENT MASK SIZE
#define PW (TILE_WIDTH + FILTERDIM - 1 )

//FILTERS=======================================================================

//3x3 filters

//Edge detection 3x3
float filter[] = {-1.0, 0.0, 1.0, -2.0, 0.0, 2.0, -1.0, 0.0, 1.0};
//Sharpening 3x3
// float filter[] = {-1.0, -1.0, -1.0, -1.0, 9, -1.0, -1.0, -1.0, -1.0};
// //blur/average 3x3
// float filter[] = {0.111, 0.111, 0.111, 0.111, 0.111, 0.111, 0.111, 0.111, 0.111};
//
// //5x5 filters
//
// //Sharpening 5x5
// float filter[] = {-1.0, -1.0, -1.0, -1.0, -1.0, -1.0, -1.0, -1.0, -1.0, -1.0,-1.0, -1.0, 25, -1.0, -1.0,-1.0, -1.0, -1.0, -1.0, -1.0,-1.0, -1.0, -1.0, -1.0, -1.0};
// //Averaging 5x5
// float filter[] = {0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04,0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04,};
//
// //7x7 Filters
//
// //sharpening 7x7
 //float filter[] = {-1.0, -1.0, -1.0, -1.0, -1.0, -1.0 ,-1.0, -1.0, -1.0, -1.0, -1.0, -1.0, -1.0 ,-1.0, -1.0, -1.0, -1.0, -1.0, -1.0, -1.0 ,-1.0,-1.0, -1.0, -1.0, 49, -1.0, -1.0 ,-1.0,- 1.0, -1.0, -1.0, -1.0, -1.0, -1.0 ,-1.0,-1.0, -1.0, -1.0, -1.0, -1.0, -1.0 ,-1.0,-1.0, -1.0, -1.0, -1.0, -1.0, -1.0 ,-1.0};
//
//
// 	//Averaging
// float filter[] = {1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0, 1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0, 1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0, 1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0, 1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0, 1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0, 1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,1.0/49.0,}
//==============================================================================



texture<float, 2, hipReadModeElementType> tex;

const char *imageFilename = "lena_bw.pgm";

const char *sampleName = "HPCAssignment2.cu";

//KERNELS ======================================================================

//Texture memory kernel=========================================================
__global__ void GPUTextureConv(float* doutput, float* filter, int width, int height, int filterDim){

		unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;           //calculate x coord
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;                       //calculate y coord

    float u = (float)x +0.5f;                                                //convert x and y coordinate, include pixel width to get center
    float v = (float)y +0.5f;

		int offset = ((filterDim-1)/2);                              //calculate offset from the filter, so if its 3x3 it will be 1
		int i = y*width +x ;                                       //calculate index in the 1D array for output data
		float sum=0.0;                                           //initlize sum

		for(int k=0;k<filterDim;k++){
			for(int l = 0; l<filterDim;l++){
			if((u-offset+l >= 0 )&&(u-offset+l < width ) && (v-offset+k >=0) && (v-offset+k < height))  //make sure point is within image boundary
			sum+=	tex2D(tex,u-offset+l,v-offset+k)*filter[l+k*filterDim];     //fetch pixel data from texture memory and multiply by corresponding filter value
			}
		}

		if(sum<0)                    //normalize
			sum=0.0;
		if(sum>1)
			sum=1.0 ;

			doutput[i] = sum;

	}


//==============================================================================

//SHARED MEMORY KERNEL==========================================================
__global__ void GPUSharedConv(float* doutput,float* ddata,  float* filter, int imageWidth, int imageHeight, int filterDim){


	__shared__ float shared_block[PW][PW] ;  //initlise tile of image in  shared memory

	int offset = filterDim/2 ;          //calculate offset


	int i = threadIdx.y*TILE_WIDTH + threadIdx.x;      //get first set of threads index
	int iy = i/PW ;                                 //get y index for tiled shared memory
	int ix = i%PW ;                                 // get x index for tiled shared memory

	int indexY = blockIdx.y*TILE_WIDTH + iy- offset ;         //get index with reference to ddata
	int indexX = blockIdx.x*TILE_WIDTH + ix - offset ;
  int index = indexY*imageWidth + indexX ;


	if((indexY >= 0) && (indexY < imageHeight) && (indexX >=0) && (indexX < imageWidth)){ //check if index in bounds
		shared_block[iy][ix] = ddata[index];  //copy into shared memory
	}else{
		shared_block[iy][ix] = 0.0;            //pad the array,
	}

 i = threadIdx.y*TILE_WIDTH + threadIdx.x + TILE_WIDTH*TILE_WIDTH;      //this is to do the next set of arrays, since the grid size is bigger than the tle size we have to assign in 2 batches
 iy = i/PW ;                                 //get y index for tiled shared memory
 ix = i%PW ;                                 // get x index for shared memory


//get index with reference to ddata
 indexY = blockIdx.y*TILE_WIDTH + iy - offset ;
 indexX = blockIdx.x*TILE_WIDTH + ix - offset ;
 index = indexY*imageWidth + indexX ;


	if(iy < PW){             //if thread is within the shared memory space
		if((indexY >= 0) && (indexY < imageHeight) && (indexX >=0) && (indexX < imageWidth)){ //check if index in bounds
			shared_block[iy][ix] = ddata[index];  //copy into shared memory
		}else{
			shared_block[iy][ix] = 0.0;         //PAD
		}
	}

	__syncthreads() ;  //wait for threads to catch up

	float sum = 0 ;  //initialise sum variable

	int k,l;             // counting variables

	for(k= 0; k < filterDim ; k++){
		for( l=0; l <filterDim; l++){

			sum+= shared_block[threadIdx.y+k][threadIdx.x+l]*filter[k*filterDim + l]; //do the convolution
		}
	}

	int y = blockIdx.y*TILE_WIDTH + threadIdx.y;   //get y index with refernece to image
	int x = blockIdx.x*TILE_WIDTH + threadIdx.x;  //get x index with reference to image

	if(y<imageHeight && x<imageWidth){
		if(sum>1)  //threshhold function
		sum=1 ;
		if(sum<0)
		sum=0;
		doutput[y*imageWidth + x] = sum ;   //assign final value
		__syncthreads();
}

}

//==============================================================================

__constant__ float dconstantFilter[FILTERDIM*FILTERDIM];            //define array for filter in constant memory

//CONSTANT MEMORY FILTER IMPLEMENTATION=========================================
__global__ void GPUConstantConv(float* ddata, float* doutput,int imageWidth, int imageHeight, int filterDim){

	int k,l;                          //counting variables
	float sum=0.0;                          //temp sum
	int offset = ((filterDim-1)/2);     //bounds for inner loop
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;              //find x dimension
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;		           //find y dimension

	int i = y*imageWidth+ x ;                                          //find unique index for each gpu

	for(k=0; k<filterDim; k++){                                       //calculate CONVOLUTION
		for(l=0; l<filterDim ; l++){
			if((i+l-offset+(k-offset)*imageWidth >= 0) && (i+l-offset+(k-offset)*imageWidth< imageWidth*imageHeight) && (i%imageWidth + l - offset >= 0) && (i%imageWidth + l - offset < imageWidth) && (i%imageHeight + k - offset >=0) && (i%imageHeight +k - offset < imageHeight))                        //COnditions if the filter falls over the image or off. First 2 check the width and last 2 check the heights

				sum+= ddata[i+l-offset+(k-offset)*imageWidth]*dconstantFilter[l+k*filterDim] ;
			}
		}

		if(sum<0)
			sum=0;                                                     //normalise values
		if(sum>1)
			sum=1 ;
			doutput[i] = sum ;                                      //assign output

}

//==============================================================================

//NAIVE GPU IMPLEMENTATION OF CONVOLUTION=======================================

__global__ void GPUNaiveConv(float* ddata, float* doutput, float* filter, int imageWidth, int imageHeight, int filterDim){

	int k,l;                          //counting variables
	float sum=0.0;                          //temp sum
	int offset = ((filterDim-1)/2);     //bounds for inner loop

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	int i = y*imageWidth+ x ;
	for(k=0; k<filterDim; k++){
		for(l=0; l<filterDim ; l++){
			if((i+l-offset+(k-offset)*imageWidth >= 0) && (i+l-offset+(k-offset)*imageWidth< imageWidth*imageHeight) && (i%imageWidth + l - offset >= 0) && (i%imageWidth + l - offset < imageWidth) && (i%imageHeight + k - offset >=0) && (i%imageHeight +k - offset < imageHeight))                        //COnditions if the filter falls over the image or off. First 2 check the width and last 2 check the heights
				sum+= ddata[i+l-offset+(k-offset)*imageWidth]*filter[l+k*filterDim] ;
		}
	}
	if(sum<0)
		sum=0;
	if(sum>1)
		sum=1 ;

	doutput[i] = sum ;


}

//==============================================================================


//CPU IMPLEMENTATION OF CONVOLUTION=============================================

void CPUConv(float* hdata, float* houtput, float* filter, int imageWidth, int imageHeight, int filterDim){

	int i,k,l;                          //counting variables
	float sum;                          //temp sum
	int offset = floor(filterDim/2);     //bounds for inner loop

	for(i=0; i< imageWidth*imageHeight; i++){
		sum=0.0;
		for(k=0; k<filterDim; k++){
			for(l=0; l<filterDim ; l++){
				if((i+l-offset+(k-offset)*imageWidth >= 0) && (i+l-offset+(k-offset)*imageWidth< imageWidth*imageHeight) && (i%imageWidth + l - offset >= 0) && (i%imageWidth + l - offset < imageWidth) && (i%imageHeight + k - offset >=0) && (i%imageHeight +k - offset < imageHeight))                        //COnditions if the filter falls over the image or off. First 2 check the width and last 2 check the heights
					sum+= hdata[i+l-offset+(k-offset)*imageWidth]*filter[l+k*filterDim] ;
			}
		}

		if(sum<0)
			sum=0;
		if(sum>1)
			sum=1 ;
		 houtput[i] = sum ;
	}

}

//==============================================================================

//TESTS=========================================================================

//Run CPU test with timings=====================================================

void runCPUTest(float* hData, unsigned int width, unsigned int height,int filterDim,int size,float* filter){
	float *hOutputData = (float *) malloc(size);                        //create an array to store the final

	hipEvent_t cpulaunch_begin, cpulaunch_end;
	hipEventCreate(&cpulaunch_begin);
	hipEventCreate(&cpulaunch_end);
	// record a CUDA event immediately before and after the kernel launch
	hipEventRecord(cpulaunch_begin,0);

	CPUConv(hData,hOutputData,filter,width,height,filterDim);   //Run cpu version

	hipEventRecord(cpulaunch_end,0);
	hipEventSynchronize(cpulaunch_end);
	// measure the time (ms) spent in the kernel
	float cputime = 0;
	hipEventElapsedTime(&cputime, cpulaunch_begin, cpulaunch_end);
	printf("CPU run time: %fms\n", cputime);
	sdkSavePGM("Image_CPU_OUT.pgm",hOutputData,width,height);
	free(hOutputData);
}

//==============================================================================

//Run naive kernel test=========================================================

int runNaiveTest(float* hData, unsigned int width, unsigned int height,int filterDim,int size,float* filter){


		 float *hOutputData = (float *) malloc(size);
		 hipEvent_t nglaunch_begin, nglaunch_end,total_begin,total_end;

//record total time including memory addresses
		 hipEventCreate(&total_begin);
		 hipEventCreate(&total_end);
		 hipEventRecord(total_begin,0);


		hipEventCreate(&nglaunch_begin);
	 	hipEventCreate(&nglaunch_end);
	 // record a CUDA event for the entire
	 	hipEventRecord(nglaunch_begin,0);

		 float *dData = 0;
		 float *dOutput = 0;
		 float *dFilter = 0;                      //create arrays we will need to give to device

		 hipMalloc((void**)&dData, size);
	 	 hipMalloc((void**)&dOutput, size);
		 hipMalloc((void**)&dFilter, filterDim*filterDim*sizeof(float));       //assign the space required for the above arrays

		 if(dOutput == 0 || dData == 0 || dFilter == 0)                       //check if the arrays actually initialised properly
		  {
		    printf("couldn't allocate device memory\n");
		    return 1;
		  }

			hipMemcpy(dData, hData, size, hipMemcpyHostToDevice);            //Copy the image to the device
			hipMemcpy(dFilter, filter, filterDim*filterDim*sizeof(float), hipMemcpyHostToDevice);	   //copy the filter to the device

			const size_t block_size = 64;                                 //initialise block size
	  	size_t grid_size = width*height / block_size;                   // calculate gride size

	  	// deal with a possible partial final block
	 	 	if(width*height % block_size) ++grid_size;


	  	hipEventCreate(&nglaunch_begin);
	  	hipEventCreate(&nglaunch_end);
		// record a CUDA event immediately before and after the kernel launch
	  	hipEventRecord(nglaunch_begin,0);
	  // launch the kernel
	  	GPUNaiveConv<<<grid_size, block_size>>>(dData,dOutput,dFilter,width,height,filterDim) ;          //Call the kernal
			hipEventRecord(nglaunch_end,0);
	 	 	hipEventSynchronize(nglaunch_end);
		// measure the time (ms) spent in the kernel
	  	float ngtime = 0;
	  	hipEventElapsedTime(&ngtime, nglaunch_begin, nglaunch_end);

	  // copy the result back to the host memory space
	  	hipMemcpy(hOutputData, dOutput, size, hipMemcpyDeviceToHost);

			checkCudaErrors(hipFree(dData)); //fre all cuda memory used
			checkCudaErrors(hipFree(dOutput));
			checkCudaErrors(hipFree(dFilter));

			hipEventRecord(total_end,0);
			hipEventSynchronize(total_end);
		// measure the time (ms) spent in the kernel
			float totaltime = 0;
			hipEventElapsedTime(&totaltime, total_begin, total_end);


//print time results
			printf("\nGPU NAIVE TIMES =====================\n");
			printf("GPU Naive run time: %fms\n", ngtime);
			printf("GPU NAIVE Memory allocation time: %fms \n", totaltime-ngtime) ;
			printf("GPU NAIVE TOTAL TIME: %fms \n",totaltime);
			printf("===================================== \n \n");


			sdkSavePGM("Image_NAIVE_OUT.pgm",hOutputData,width,height);               //save the new image as Image_out.pgm

			free(hOutputData);


			return(0);
}

//==============================================================================

//run shared memory kernel test=================================================

int runSharedTest(float* hData, unsigned int width, unsigned int height,int filterDim,int size,float* filter){

	hipEvent_t cslaunch_begin, cslaunch_end,total_begin,total_end;

	hipEventCreate(&total_begin);
	hipEventCreate(&total_end);
	hipEventRecord(total_begin,0);

 	float *hsOutputData = (float *) malloc(size);

 	float *dsData = 0;
 	float *dsOutput = 0;
 	float *dsFilter = 0;

 	hipMalloc((void**)&dsData, size);
 	hipMalloc((void**)&dsOutput, size);
 	hipMalloc((void**)&dsFilter, filterDim*filterDim*sizeof(float));       //assign the space required for the above arrays


 	if(dsOutput == 0 || dsData == 0 || dsFilter == 0)                       //check if the arrays actually initialised properly
 	{
 		printf("couldn't allocate device memory (shared)\n");
 		return 1;
 	}

 	int offset = ((filterDim-1)/2);

 	 dim3 sdimGrid(ceil((float) width/TILE_WIDTH), ceil((float) height/TILE_WIDTH));				//initlise block and grid size, 2D
 	 dim3 sdimBlock(TILE_WIDTH,TILE_WIDTH);


  checkCudaErrors(hipMemcpy(dsData, hData, size, hipMemcpyHostToDevice));            //Copy the image to the device
  checkCudaErrors(hipMemcpy(dsFilter, filter, filterDim*filterDim*sizeof(float), hipMemcpyHostToDevice));	   //copy the filter to the device

  hipEventCreate(&cslaunch_begin);
  hipEventCreate(&cslaunch_end);
  // record a CUDA event immediately before and after the kernel launch
  hipEventRecord(cslaunch_begin,0);
  // launch the kernel
  GPUSharedConv<<<sdimGrid, sdimBlock>>>(dsOutput,dsData,dsFilter, width, height,filterDim);

  hipEventRecord(cslaunch_end,0);
  hipEventSynchronize(cslaunch_end);
  // measure the time (ms) spent in the kernel
 		float cstime =0;
 	//	printf("%f ", hData[3]);
 		hipEventElapsedTime(&cstime, cslaunch_begin, cslaunch_end);

  // copy the result back to the host memory space
  hipMemcpy(hsOutputData, dsOutput, size, hipMemcpyDeviceToHost);

	checkCudaErrors(hipFree(dsData));
	checkCudaErrors(hipFree(dsOutput));
	checkCudaErrors(hipFree(dsFilter));


//check total time for hipArray
	hipEventRecord(total_end,0);
	hipEventSynchronize(total_end);
// measure the time (ms) spent in the kernel
	float totaltime = 0;
	hipEventElapsedTime(&totaltime, total_begin, total_end);


//print time results
	printf("GPU Shared TIMES =====================\n");
	printf("GPU Shared run time: %fms\n", cstime);
	printf("GPU Shared Memory allocation time: %fms \n", totaltime-cstime) ;
	printf("GPU Shared TOTAL TIME: %fms \n",totaltime);
	printf("===================================== \n \n");

  sdkSavePGM("Image_SHARED_OUT.pgm",hsOutputData,width,height);
	free(hsOutputData);

	return(0);

}

//==============================================================================

//run constant memory kernel test===============================================

int runConstantTest(float* hData, unsigned int width, unsigned int height,int filterDim,int size,float* filter){
		hipEvent_t cglaunch_begin, cglaunch_end,total_end,total_begin;

		//record total time including memory addresses
		hipEventCreate(&total_begin);
		hipEventCreate(&total_end);
		hipEventRecord(total_begin,0);

		float *hOutputData = (float *) malloc(size);
		float *dcData = 0;
		float *dcOutput = 0;
		float *dcFilter = 0;

		hipMalloc((void**)&dcData, size);
		hipMalloc((void**)&dcOutput, size);
		hipMalloc((void**)&dcFilter, filterDim*filterDim*sizeof(float));       //assign the space required for the above arrays
		hipMalloc((void**)&dconstantFilter, filterDim*filterDim*sizeof(float));       //assign the space required for the above arrays

	 if(dcOutput == 0 || dcData == 0 || dcFilter == 0)                       //check if the arrays actually initialised properly
		{
			printf("couldn't allocate device memory (constant)\n");
			return 1;
		}

		hipMemcpy(dcData, hData, size, hipMemcpyHostToDevice);            //Copy the image to the device
		hipMemcpy(dcFilter, filter, filterDim*filterDim*sizeof(float), hipMemcpyHostToDevice);	   //copy the filter to the device
		hipMemcpyToSymbol(HIP_SYMBOL(dconstantFilter),filter,sizeof(float)*filterDim*filterDim,0,hipMemcpyHostToDevice) ;


		const size_t block_size = 64;                                 //initialise block size                  // calculate gride size
		size_t cgrid_size = width*height / block_size;                   // calculate gride size

	  	// deal with a possible partial final block
	 	 if(width*height % block_size) ++cgrid_size;

		 hipEventCreate(&cglaunch_begin);
		 hipEventCreate(&cglaunch_end);
		 // record a CUDA event immediately before and after the kernel launch
		 hipEventRecord(cglaunch_begin,0);
		 // launch the kernel
		 GPUConstantConv<<<cgrid_size, block_size>>>(dcData,dcOutput,width,height,filterDim) ;          //Call the kernal
		 hipEventRecord(cglaunch_end,0);
		 hipEventSynchronize(cglaunch_end);
		 // measure the time (ms) spent in the kernel
		 		float cgtime =0;
		 		hipEventElapsedTime(&cgtime, cglaunch_begin, cglaunch_end);

		 // copy the result back to the host memory space
		 hipMemcpy(hOutputData, dcOutput, size, hipMemcpyDeviceToHost);

		 checkCudaErrors(hipFree(dcData));
		 checkCudaErrors(hipFree(dcOutput));
		 checkCudaErrors(hipFree(dcFilter));

		 hipEventRecord(total_end,0);
		 hipEventSynchronize(total_end);
	 // measure the time (ms) spent in the kernel
		 float totaltime = 0;
		 hipEventElapsedTime(&totaltime, total_begin, total_end);

	//print time results
		 printf("GPU Constant TIMES =====================\n");
		 printf("GPU Constant run time: %fms\n", cgtime);
		 printf("GPU Constant Memory allocation time: %fms \n", totaltime-cgtime) ;
		 printf("GPU Constant TOTAL TIME %fms: \n",totaltime);
		 printf("===================================== \n \n");


  	sdkSavePGM("Image_NAIVE_OUT.pgm",hOutputData,width,height);

		 sdkSavePGM("Image_CONST_OUT.pgm",hOutputData,width,height);
		 free(hOutputData);

		 return(0);

}

//==============================================================================

//run texture memory kernel test================================================
int runTextureTest(float* hData, unsigned int width, unsigned int height,int filterDim,int size,float* filter){
		hipEvent_t tglaunch_begin, tglaunch_end,total_end,total_begin;

		//record total time including memory addresses
		hipEventCreate(&total_begin);
		hipEventCreate(&total_end);
		hipEventRecord(total_begin,0);


		float *dtFilter = 0;                               //create pointer for filter
		float *dData = 0 ;
		float *hOutputData = (float *) malloc(size);


		 hipMalloc((void**)&dtFilter, filterDim*filterDim*sizeof(float)); 				//allocate memory for filter
		 hipMemcpy(dtFilter, filter, filterDim*filterDim*sizeof(float), hipMemcpyHostToDevice);	   //copy the filter to the device

		 checkCudaErrors(hipMalloc((void **) &dData, size));

		 // Allocate array and copy image data
		 hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
		 hipArray *cuArray;
		 checkCudaErrors(hipMallocArray(&cuArray, &channelDesc, width, height));
		 checkCudaErrors(hipMemcpyToArray(cuArray,0,0, hData,size, hipMemcpyHostToDevice));

		 // Set texture parameters
		 tex.addressMode[0] = hipAddressModeWrap;
		 tex.addressMode[1] = hipAddressModeWrap;
		 tex.filterMode = hipFilterModeLinear;
		 tex.normalized = false;    // access with normalized texture coordinates

		 // Bind the array to the texture
		 checkCudaErrors(hipBindTextureToArray(tex, cuArray, channelDesc));

		 dim3 dimBlock(8, 8, 1);				//initlise block and grid size, 2D
		 dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

		 checkCudaErrors(hipDeviceSynchronize());       //wait for all threads
		 // Execute the kernel

		 hipEventCreate(&tglaunch_begin);                   //start timer
		 hipEventCreate(&tglaunch_end);
		 // record a CUDA event immediately before and after the kernel launch
		 hipEventRecord(tglaunch_begin,0);
		 GPUTextureConv<<<dimGrid, dimBlock, 0>>>(dData,dtFilter, width, height,filterDim);

		 // Check if kernel execution generated an error
		 getLastCudaError("Kernel execution failed");
		 hipEventRecord(tglaunch_end,0);
		 hipEventSynchronize(tglaunch_end);
		 // measure the time (ms) spent in the kernel
		 float tgtime =0;
		 hipEventElapsedTime(&tgtime, tglaunch_begin, tglaunch_end);


		 // copy result from device to host
		 checkCudaErrors(hipMemcpy(hOutputData,dData,size,	hipMemcpyDeviceToHost));

		 //free device memory
		 checkCudaErrors(hipFree(dData));
		 checkCudaErrors(hipFree(dtFilter));


		 hipEventRecord(total_end,0);
		 hipEventSynchronize(total_end);
	 // measure the time (ms) spent in the kernel
		 float totaltime = 0;
		 hipEventElapsedTime(&totaltime, total_begin, total_end);


//print time results
		 printf("GPU Texture TIMES =====================\n");
		 printf("GPU Texture run time: %fms: \n", tgtime);
		 printf("GPU Texture Memory allocation time: %fms \n", totaltime-tgtime) ;
		 printf("GPU Texture TOTAL TIME %fms: \n",totaltime);
		 printf("===================================== \n \n");

		 sdkSavePGM("Image_TEXT_OUT.pgm",hOutputData,width,height);

		 free(hOutputData);

		 return(0);

}
//==============================================================================

int main(int argc, char **argv){

size_t available, total;
hipMemGetInfo(&available,&total);



	int devID = findCudaDevice(argc, (const char **) argv);
    // load image from disk
    	float *hData = NULL;                                                 //To store the image
    	unsigned int width, height;
    	char *imagePath = sdkFindFilePath(imageFilename, argv[0]);           //find the image path

//DEFINE FILTER HERE============================================================
//	float filter[] ={0.0,0.0,0.0,0.0,1.0,0.0,0.0,0.0,0.0};                 //returns original image
//	 float filter[] ={-1.0,0.0,1.0,-2.0,0.0,2.0,-1.0,0.0,1.0};            //highlights edges
//	float filter[] ={-1.0,-1.0,-1.0,-1.0,9.0,-1.0,-1.0,-1.0,-1.0};      //sharpens image
//	float filter[] ={1.0/9.0,1.0/9.0,1.0/9.0,1.0/9.0,1.0/9.0,1.0/9.0,1.0/9.0,1.0/9.0,1.0/9.0};   //slightly blurs image

	int filterDim=FILTERDIM;  						//dimensions of the filter, assume its square
//==============================================================================

    	if (imagePath == NULL)
    	{
       	 printf("Unable to source image file: %s\n", imageFilename);
       	 exit(EXIT_FAILURE);
    	}

    	sdkLoadPGM(imagePath, &hData, &width, &height);             //load image into hdata and initialize width and height, hdata is a 1D array
    	unsigned int size = width * height * sizeof(float);                 //get total size of image (length of array) in bytes


			runCPUTest(hData,width,height,filterDim,size,filter);
			runNaiveTest(hData,width,height,filterDim,size,filter);
			// runConstantTest(hData,width,height,filterDim,size,filter);
			runConstantTest(hData,width,height,filterDim,size,filter);

			runTextureTest(hData,width,height,filterDim,size,filter);
			runSharedTest(hData,width,height,filterDim,size,filter);


}
