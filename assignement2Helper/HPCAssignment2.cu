#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <./helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#define MAX_EPSILON_ERROR 5e-3f
#define TILE_WIDTH 16
#define FILTERDIM 3                     //CHANGE THIS WHEN USING DIFFERENT MASK SIZE
#define PW (TILE_WIDTH + FILTERDIM - 1 )

texture<float, 2, hipReadModeElementType> tex;

const char *imageFilename = "lena_bw.pgm";

const char *sampleName = "HPCAssignment2.cu";


//Texture memory kernel=========================================================
__global__ void GPUTextureConv(float* doutput, float* filter, int width, int height, int filterDim){

		unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;           //calculate x coord
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;                       //calculate y coord

    float u = (float)x +0.5f;                                                //convert x and y coordinate, include pixel width to get center
    float v = (float)y +0.5f;

		int offset = ((filterDim-1)/2);                              //calculate offset from the filter, so if its 3x3 it will be 1
		int i = y*width +x ;                                       //calculate index in the 1D array for output data
		float sum=0.0;                                           //initlize sum

		for(int k=0;k<filterDim;k++){
			for(int l = 0; l<filterDim;l++){
			if((u-offset+l >= 0 )&&(u-offset+l < width ) && (v-offset+k >=0) && (v-offset+k < height))  //make sure point is within image boundary
			sum+=	tex2D(tex,u-offset+l,v-offset+k)*filter[l+k*filterDim];     //fetch pixel data from texture memory and multiply by corresponding filter value
			}
		}

		if(sum<0)                    //normalize
			sum=0;
		if(sum>1)
			sum=1 ;
			doutput[i] = sum;
	}


//==============================================================================


//SHARED MEMORY TILING IMPLEMENTATION===========================================
//do later
__global__ void GPUSharedConv(float* ddata, float* doutput, float* filter,int blockWidth, int imageWidth, int imageHeight, int filterDim){





}


__constant__ float dconstantFilter[FILTERDIM*FILTERDIM];            //define array for filter in constant memory
//CONSTANT MEMORY FILTER IMPLEMENTATION=========================================
__global__ void GPUConstantConv(float* ddata,const float *__restrict__ kernel, float* doutput,int imageWidth, int imageHeight, int filterDim){

	int k,l;                          //counting variables
	float sum=0.0;                          //temp sum
	int offset = ((filterDim-1)/2);     //bounds for inner loop

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;              //find x dimension
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;		           //find y dimension

	int i = y*imageWidth+ x ;                                          //find unique index for each gpu

	for(k=0; k<filterDim; k++){                                       //calculate CONVOLUTION
		for(l=0; l<filterDim ; l++){
			if((i+l-offset+(k-offset)*imageWidth >= 0) && (i+l-offset+(k-offset)*imageWidth< imageWidth*imageHeight) && (i%imageWidth + l - offset >= 0) && (i%imageWidth + l - offset < imageWidth) && (i%imageHeight + k - offset >=0) && (i%imageHeight +k - offset < imageHeight))                        //COnditions if the filter falls over the image or off. First 2 check the width and last 2 check the heights

				sum+= ddata[i+l-offset+(k-offset)*imageWidth]*dconstantFilter[l+k*filterDim] ;
			}
		}

		if(sum<0)
			sum=0;                                                     //normalise values
		if(sum>1)
			sum=1 ;
			doutput[i] = sum ;                                      //assign output

}

//==============================================================================

//==============================================================================

//NAIVE GPU IMPLEMENTATION OF CONVOLUTION=======================================
__global__ void GPUNaiveConv(float* ddata, float* doutput, float* filter, int imageWidth, int imageHeight, int filterDim){

	int k,l;                          //counting variables
	float sum=0.0;                          //temp sum
	int offset = ((filterDim-1)/2);     //bounds for inner loop

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	int i = y*imageWidth+ x ;

	for(k=0; k<filterDim; k++){
		for(l=0; l<filterDim ; l++){
			if((i+l-offset+(k-offset)*imageWidth >= 0) && (i+l-offset+(k-offset)*imageWidth< imageWidth*imageHeight) && (i%imageWidth + l - offset >= 0) && (i%imageWidth + l - offset < imageWidth) && (i%imageHeight + k - offset >=0) && (i%imageHeight +k - offset < imageHeight))                        //COnditions if the filter falls over the image or off. First 2 check the width and last 2 check the heights
				sum+= ddata[i+l-offset+(k-offset)*imageWidth]*filter[l+k*filterDim] ;
		}
	}
	if(sum<0)
		sum=0;
	if(sum>1)
		sum=1 ;

	doutput[i] = sum ;


}
//==============================================================================


//CPU IMPLEMENTATION OF CONVOLUTION=============================================
void CPUConv(float* hdata, float* houtput, float* filter, int imageWidth, int imageHeight, int filterDim){

	int i,k,l;                          //counting variables
	float sum;                          //temp sum
	int offset = floor(filterDim/2);     //bounds for inner loop
//	printf("%d \n",imageWidth*imageHeight);
	for(i=0; i< imageWidth*imageHeight; i++){
		sum=0.0;
		for(k=0; k<filterDim; k++){
			for(l=0; l<filterDim ; l++){
				if((i+l-offset+(k-offset)*imageWidth >= 0) && (i+l-offset+(k-offset)*imageWidth< imageWidth*imageHeight) && (i%imageWidth + l - offset >= 0) && (i%imageWidth + l - offset < imageWidth) && (i%imageHeight + k - offset >=0) && (i%imageHeight +k - offset < imageHeight))                        //COnditions if the filter falls over the image or off. First 2 check the width and last 2 check the heights
					sum+= hdata[i+l-offset+(k-offset)*imageWidth]*filter[l+k*filterDim] ;
			}
		}

		if(sum<0)
			sum=0;
		if(sum>1)
			sum=1 ;
		houtput[i] = sum ;
	}

}

//==============================================================================

int main(int argc, char **argv){

	int devID = findCudaDevice(argc, (const char **) argv);
    // load image from disk
    	float *hData = NULL;                                                 //To store the image
    	unsigned int width, height;
    	char *imagePath = sdkFindFilePath(imageFilename, argv[0]);           //find the image path

//DEFINE FILTER HERE============================================================
//	float filter[] ={0.0,0.0,0.0,0.0,1.0,0.0,0.0,0.0,0.0};                 //returns original image
	 float filter[] ={-1.0,0.0,1.0,-2.0,0.0,2.0,-1.0,0.0,1.0};            //highlights edges
//	float filter[] ={-1.0,-1.0,-1.0,-1.0,9.0,-1.0,-1.0,-1.0,-1.0};      //sharpens image
//	float filter[] ={1.0/9.0,1.0/9.0,1.0/9.0,1.0/9.0,1.0/9.0,1.0/9.0,1.0/9.0,1.0/9.0,1.0/9.0};   //slightly blurs image

	int filterDim=FILTERDIM;  						//dimensions of the filter, assume its square
//==============================================================================

    	if (imagePath == NULL)
    	{
       	 printf("Unable to source image file: %s\n", imageFilename);
       	 exit(EXIT_FAILURE);
    	}

    	sdkLoadPGM(imagePath, &hData, &width, &height);             //load image into hdata and initialize width and height, hdata is a 1D array
			//printf("%d %d \n", width,height);
    	unsigned int size = width * height * sizeof(float);                 //get total size of image (length of array) in bytes
    	float *hOutputData = (float *) malloc(size);                        //create an array to store the final


//RUN CPU VERSION ==============================================================
		hipEvent_t cpulaunch_begin, cpulaunch_end;
		hipEventCreate(&cpulaunch_begin);
		hipEventCreate(&cpulaunch_end);
// record a CUDA event immediately before and after the kernel launch
		hipEventRecord(cpulaunch_begin,0);

 	  CPUConv(hData,hOutputData,filter,width,height,filterDim);   //Run cpu version

		hipEventRecord(cpulaunch_end,0);
 		hipEventSynchronize(cpulaunch_end);
// measure the time (ms) spent in the kernel
	 float cputime = 0;
	 hipEventElapsedTime(&cputime, cpulaunch_begin, cpulaunch_end);
	 printf("CPU run time: %fms\n", cputime);
	 sdkSavePGM("Image_CPU_OUT.pgm",hOutputData,width,height);
//==============================================================================

 //Run Naive GPU VERSION========================================================
	 hipEvent_t nglaunch_begin, nglaunch_end;

	 float *dData = 0;
	 float *dOutput = 0;
	 float *dFilter = 0;                      //create arrays we will need to give to device

	 hipMalloc((void**)&dData, size);
 	 hipMalloc((void**)&dOutput, size);
	 hipMalloc((void**)&dFilter, filterDim*filterDim*sizeof(float));       //assign the space required for the above arrays

	 if(dOutput == 0 || dData == 0 || dFilter == 0)                       //check if the arrays actually initialised properly
	  {
	    printf("couldn't allocate device memory\n");
	    return 1;
	  }

		hipMemcpy(dData, hData, size, hipMemcpyHostToDevice);            //Copy the image to the device
		hipMemcpy(dFilter, filter, filterDim*filterDim*sizeof(float), hipMemcpyHostToDevice);	   //copy the filter to the device

		const size_t block_size = 64;                                 //initialise block size
  	size_t grid_size = width*height / block_size;                   // calculate gride size

  	// deal with a possible partial final block
 	 	if(width*height % block_size) ++grid_size;


  	hipEventCreate(&nglaunch_begin);
  	hipEventCreate(&nglaunch_end);
	// record a CUDA event immediately before and after the kernel launch
  	hipEventRecord(nglaunch_begin,0);
  // launch the kernel
  	GPUNaiveConv<<<grid_size, block_size>>>(dData,dOutput,dFilter,width,height,filterDim) ;          //Call the kernal
		hipEventRecord(nglaunch_end,0);
 	 	hipEventSynchronize(nglaunch_end);
	// measure the time (ms) spent in the kernel
  	float ngtime = 0;
  	hipEventElapsedTime(&ngtime, nglaunch_begin, nglaunch_end);

  // copy the result back to the host memory space
  	hipMemcpy(hOutputData, dOutput, size, hipMemcpyDeviceToHost);
		printf("GPU Naive run time: %fms\n", ngtime);

		sdkSavePGM("Image_NAIVE_OUT.pgm",hOutputData,width,height);               //save the new image as Image_out.pgm
//==============================================================================

//RUN THE CONSTANT MEMORY GPU implementation====================================
	hipEvent_t cglaunch_begin, cglaunch_end;

	float *dcData = 0;
	float *dcOutput = 0;
	float *dcFilter = 0;

	hipMalloc((void**)&dcData, size);
	hipMalloc((void**)&dcOutput, size);
	hipMalloc((void**)&dcFilter, filterDim*filterDim*sizeof(float));       //assign the space required for the above arrays
	hipMalloc((void**)&dconstantFilter, filterDim*filterDim*sizeof(float));       //assign the space required for the above arrays

 if(dcOutput == 0 || dcData == 0 || dcFilter == 0)                       //check if the arrays actually initialised properly
	{
		printf("couldn't allocate device memory (constant)\n");
		return 1;
	}

	hipMemcpy(dcData, hData, size, hipMemcpyHostToDevice);            //Copy the image to the device
	hipMemcpy(dcFilter, filter, filterDim*filterDim*sizeof(float), hipMemcpyHostToDevice);	   //copy the filter to the device
	hipMemcpyToSymbol(HIP_SYMBOL(dconstantFilter),filter,sizeof(float)*filterDim*filterDim,0,hipMemcpyHostToDevice) ;



	size_t cgrid_size = width*height / block_size;                   // calculate gride size

  	// deal with a possible partial final block
 	 if(width*height % block_size) ++cgrid_size;

	 hipEventCreate(&cglaunch_begin);
	 hipEventCreate(&cglaunch_end);
	 // record a CUDA event immediately before and after the kernel launch
	 hipEventRecord(cglaunch_begin,0);
	 // launch the kernel
	 GPUConstantConv<<<cgrid_size, block_size>>>(dcData,dconstantFilter,dcOutput,width,height,filterDim) ;          //Call the kernal
	 hipEventRecord(cglaunch_end,0);
	 hipEventSynchronize(cglaunch_end);
	 // measure the time (ms) spent in the kernel
	 		float cgtime =0;
	 		hipEventElapsedTime(&cgtime, cglaunch_begin, cglaunch_end);

	 // copy the result back to the host memory space
	 hipMemcpy(hOutputData, dcOutput, size, hipMemcpyDeviceToHost);
	 printf("GPU Const run time: %fms\n", cgtime);

	 sdkSavePGM("Image_CONST_OUT.pgm",hOutputData,width,height);

//==============================================================================

//TEXTURE MEMORY IMPLEMENTATION ================================================
//float *dData = NULL;
	hipEvent_t tglaunch_begin, tglaunch_end;
	float *dtFilter = 0;                               //create pointer for filter

	 hipMalloc((void**)&dtFilter, filterDim*filterDim*sizeof(float)); 				//allocate memory for filter
	 hipMemcpy(dtFilter, filter, filterDim*filterDim*sizeof(float), hipMemcpyHostToDevice);	   //copy the filter to the device

	 checkCudaErrors(hipMalloc((void **) &dData, size));

	 // Allocate array and copy image data
	 hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	 hipArray *cuArray;
	 checkCudaErrors(hipMallocArray(&cuArray, &channelDesc, width, height));
	 checkCudaErrors(hipMemcpyToArray(cuArray,0,0, hData,size, hipMemcpyHostToDevice));

	 // Set texture parameters
	 tex.addressMode[0] = hipAddressModeWrap;
	 tex.addressMode[1] = hipAddressModeWrap;
	 tex.filterMode = hipFilterModeLinear;
	 tex.normalized = false;    // access with normalized texture coordinates

	 // Bind the array to the texture
	 checkCudaErrors(hipBindTextureToArray(tex, cuArray, channelDesc));

	 dim3 dimBlock(8, 8, 1);				//initlise block and grid size, 2D
	 dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

	 checkCudaErrors(hipDeviceSynchronize());       //wait for all threads
	 // Execute the kernel

	 hipEventCreate(&tglaunch_begin);                   //start timer
	 hipEventCreate(&tglaunch_end);
	 // record a CUDA event immediately before and after the kernel launch
	 hipEventRecord(tglaunch_begin,0);
	 GPUTextureConv<<<dimGrid, dimBlock, 0>>>(dData,dtFilter, width, height,filterDim);

	 // Check if kernel execution generated an error
	 getLastCudaError("Kernel execution failed");
	 hipEventRecord(tglaunch_end,0);
	 hipEventSynchronize(tglaunch_end);
	 // measure the time (ms) spent in the kernel
	 float tgtime =0;
	 hipEventElapsedTime(&tgtime, tglaunch_begin, tglaunch_end);


	 // copy result from device to host
	 checkCudaErrors(hipMemcpy(hOutputData,dData,size,	hipMemcpyDeviceToHost));

	 printf("GPU Texture run time: %fms\n", tgtime);

	 sdkSavePGM("Image_TEXT_OUT.pgm",hOutputData,width,height);

//==============================================================================


/*//Shared memory implementation of convolution===================================
hipEvent_t cslaunch_begin, cslaunch_end;

float *dsData = 0;
float *dsOutput = 0;
float *dsFilter = 0;

hipMalloc((void**)&dsData, size);
hipMalloc((void**)&dsOutput, size);
hipMalloc((void**)&dsFilter, filterDim*filterDim*sizeof(float));       //assign the space required for the above arrays


if(dsOutput == 0 || dsData == 0 || dsFilter == 0)                       //check if the arrays actually initialised properly
{
	printf("couldn't allocate device memory (shared)\n");
	return 1;
}

int offset = ((filterDim-1)/2);

dim3 sdimBlock(TILE_WIDTH+2*offset, TILE_WIDTH+2*offset, 1);				//initlise block and grid size, 2D
dim3 sdimGrid(width / dimBlock.x, height / dimBlock.y, 1);


 hipMemcpy(dsData, hData, size, hipMemcpyHostToDevice);            //Copy the image to the device
 hipMemcpy(dsFilter, filter, filterDim*filterDim*sizeof(float), hipMemcpyHostToDevice);	   //copy the filter to the device

 hipEventCreate(&cslaunch_begin);
 hipEventCreate(&cslaunch_end);
 // record a CUDA event immediately before and after the kernel launch
 hipEventRecord(cslaunch_begin,0);
 // launch the kernel
 GPUSharedConv<<<sdimGrid, sdimBlock, 0>>>(dsOutput,dsData,dsFilter, width, height,filterDim);
 hipEventRecord(cslaunch_end,0);
 hipEventSynchronize(cslaunch_end);
 // measure the time (ms) spent in the kernel
		float cstime =0;
		hipEventElapsedTime(&cstime, cslaunch_begin, cslaunch_end);

 // copy the result back to the host memory space
 hipMemcpy(hOutputData, dsOutput, size, hipMemcpyDeviceToHost);
 printf("GPU Shared run time: %fms\n", cstime);

 sdkSavePGM("Image_SHARED_OUT.pgm",hOutputData,width,height);*/

//==============================================================================


//	sdkSavePGM("Image_OUT.pgm",hOutputData,width,height);               //save the new image as Image_out.pgm


}
